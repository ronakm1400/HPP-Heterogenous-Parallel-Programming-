#include<stdio.h>
#include<hip/hip_runtime.h>

void printCUDADeviceProperties(void)
{

	//Code
	
	printf("CUDA Information : \n");
	printf("=======================================================================================\n");
	
	hipError_t ret_cuda_rt;
	int dev_count = 0,iCnt = 0;
	
	ret_cuda_rt = hipGetDeviceCount(&dev_count);
	if(ret_cuda_rt != hipSuccess)
	{
		printf("CUDA Runtime API Error - hipGetDeviceCount() Failed Due to %s.\n",hipGetErrorString(ret_cuda_rt));
	}
	else if(dev_count == 0)
	{
		printf("There is no CUDA supported device on this system.\n");	
	}
	else
	{
		printf("Total number of CUDA Supporting GPU Device/Devices on this system : %d\n",dev_count);
		
		for(iCnt = 0; iCnt < dev_count; iCnt++)
		{
			//Structure
			hipDeviceProp_t dev_prop;
			int driverVersion = 0,runtimeVersion = 0;
			
			ret_cuda_rt = hipGetDeviceProperties(&dev_prop,iCnt);
			if(ret_cuda_rt != hipSuccess)
			{
				printf("%s in %s at line %d\n",hipGetErrorString(ret_cuda_rt),__FILE__,__LINE__);
				return;
			}
			printf("\n");
			hipDriverGetVersion(&driverVersion);
			hipRuntimeGetVersion(&runtimeVersion);

			printf("\n=======================================================================\n");			
			printf("***************** CUDA DRIVER AND RUNTIME INFORMATION *****************\n");
			printf("=======================================================================\n");
			printf("CUDA Driver Version 				: %d.%d\n",(driverVersion/1000),((driverVersion % 100)/10));
			printf("CUDA Runtime Version 				: %d.%d\n",(runtimeVersion/1000),(runtimeVersion % 100)/10);
			printf("\n");
			
			printf("\n=======================================================================\n");
			printf("***************** GPU DEVICE GENERAL INFORMATION *****************\n");
			printf("=======================================================================\n");
			
			printf("\nGPU Device Number				: %d\n",iCnt);
			printf("GPU Device Compute Compatibility		: %d.%d\n",dev_prop.major,dev_prop.minor);
			
			printf("GPU Device Clock Rate 				: %d\n",dev_prop.clockRate);
			printf("GPU Device Type					:");
			if(dev_prop.integrated)
			{
				printf(" Integrated ( On-Board )\n");
			}	
			else
			{
				printf(" Discrete ( Card )\n");
			}
			printf("\n");
	
			printf("***************** GPU DEVICE MEMORY INFORMATION *****************\n");
			printf("=================================================================\n");
			
			printf("\nGPU Device Total Memory			: %.0fGB = %.0f MB = %llu Bytes\n",((float)dev_prop.totalGlobalMem/1048576.0f)/1024.0f,((float)dev_prop.totalGlobalMem/1048576.0f),(unsigned long long)dev_prop.totalGlobalMem);
			
			printf("GPU Device Constant Memory			: %lu Bytes\n",(unsigned long)dev_prop.totalConstMem);
			printf("GPU Device Shared Memory Per SMProcessor	: %lu \n",(unsigned long)dev_prop.sharedMemPerBlock);
			printf("\n");
			
			printf("***************** GPU DEVICE MULTIPROCESSOR INFORMATION *****************\n");
			printf("=========================================================================\n");
			
			printf("\nGPU Device Number of SMProcessors		: %d\n",dev_prop.multiProcessorCount);
			printf("GPU Device Number of Registers Per SMProcessor	: %d\n",dev_prop.regsPerBlock);
			printf("\n");				
		
			printf("***************** GPU DEVICE THREAD INFORMATION *****************\n");
			printf("=================================================================\n");
			
			printf("\nGPU Device Maximum Number Of Threads Per SMProcessor	: %d\n",dev_prop.maxThreadsPerMultiProcessor);
			printf("\nGPU Device Maximum Number Of Threads Per Block		: %d\n",dev_prop.maxThreadsPerBlock);
			printf("\nGPU Device Threads In Warp				: %d\n",dev_prop.warpSize);
			printf("\nGPU Device Maximum Thread Dimensions			: %d , %d , %d\n",dev_prop.maxThreadsDim[0],dev_prop.maxThreadsDim[1],dev_prop.maxThreadsDim[2]);
			printf("\nGPU Device Maximum Grid Dimensions			: %d , %d , %d\n",dev_prop.maxGridSize[0],dev_prop.maxGridSize[1],dev_prop.maxGridSize[2]);
			printf("\n");
			
			printf("***************** GPU DEVICE DRIVER INFORMATION *****************\n");
			printf("=================================================================\n");
			
			printf("\nGPU Device Has ECC Support 			: %s\n",dev_prop.ECCEnabled ? "Enabled" : "Disabled");
			
#if defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
			
			printf("GPU Device CUDA Driver Mode (TCC or WDDM)	: %s\n",dev_prop.tccDriver ? "TCC(Tesla Compute Cluster Driver)" : "WDDM(Windows Display Driver Model)");

#endif	
	printf("=======================================================================================\n");
						
		}	
	}
}

int main(void)
{
	
	void printCUDADeviceProperties(void);
	
	printCUDADeviceProperties();

	exit(EXIT_SUCCESS);
	
}
